#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <helper_string.h>
#include "cuda_kernels.hpp"
#include <thrust/copy.h>
#include <thrust/device_ptr.h>
#include <thrust/remove.h>
#include "KdBlock.hpp"

#include <iostream>
#include <cstdio>
#include <cstdlib>

static const int THREAD_N = 128;
static const int RECORDS_PER_THREAD = 8;

namespace mongo {

	__global__ static void cudaprint_gpu(uint64_t *tmp) {
//		tmp[0] = 4321;
		printf("In gpu \n");
//		uint64_t *st = tmp + 4;
//		printf("%lf %lf %lu %lu\n", uint2double(st[0]), uint2double(st[1]), uint2long(st[2]), st[3]);
		double * dd = (double *)tmp;
		printf("%lf %lf\n", dd[0], dd[1]);
		printf("%lf %lf\n", dd[2], dd[3]);
		printf("%lf %lf\n", dd[4], dd[5]);
	}
	
	__global__ static void cudaTestBlock(uint64_t *blockRanges, int numBlocks, uint64_t *output, uint64_t *queryRanges, uint32_t size)
	{
		const uint32_t index = (blockIdx.x * blockDim.x + threadIdx.x);
		bool matched = (index < numBlocks) && KdQuery::rangeMatched(blockRanges + index * size * 2, queryRanges, size);
		output[index] = matched ? index : 0xFFFFFFFFFFFFFFFF;
	}
	
//	__global__ static void cudaTestKey(int numRecords, TripKey *keys, uint32_t *blockIndex, KdQuery query, int nSrc, float *srcRegion, int nDst, float *dstRegion, int *output)
	__global__ static void cudaTestKey(int numRecords, TripKey *keys, uint64_t *blockIndex, uint32_t qsize, uint64_t * queryRange, int noRegions, uint32_t* regionSize, 
						uint32_t* regionOffset, float *regions, long *output)
	{
		const int tid = (blockIdx.x * blockDim.x + threadIdx.x);
		const int stride = gridDim.x * blockDim.x;
		
		for (int i = tid; i < numRecords; i += stride) {
			int offset = KdBlock::MAX_RECORDS_PER_BLOCK
					* blockIndex[i / KdBlock::MAX_RECORDS_PER_BLOCK]
					+ i % KdBlock::MAX_RECORDS_PER_BLOCK;
			offset = offset * (qsize + 1);
			bool matched = KdQuery::queryMatched(queryRange,keys + offset, qsize);
			if(matched) {
				for(int j = 0;j < noRegions;j ++) {
					matched = Neighborhoods::isInside(regionSize[j], regions + regionOffset[j], uint2double(keys[offset + j * 2 + 0]), uint2double(keys[offset + j * 2 + 1]));
					if(!matched) {
						break;
					}
				}
			}
			output[i] = matched ? (long) keys[offset + qsize] : -1;
		}
	}

	
//	__global__ static void cudaTestKeyAll(int numRecords, TripKey *keys, KdQuery query, int nSrc, float *srcRegion, int nDst, float *dstRegion, int *output)
	__global__ static void cudaTestKeyAll(int numRecords, TripKey *keys, uint32_t qsize, uint64_t * queryRange, int noRegions, uint32_t* regionSize, 
						uint32_t* regionOffset, float *regions, long *output)
	{
		const int tid = (blockIdx.x * blockDim.x + threadIdx.x);
		const int stride = gridDim.x * blockDim.x;
		for (int i = tid; i < numRecords; i += stride) {
			int offset = i * (qsize + 1);
			bool matched = KdQuery::queryMatched(queryRange,keys + offset, qsize);
			if(matched) {
				for(int j = 0;j < noRegions;j ++) {
					matched = Neighborhoods::isInside(regionSize[j], regions + regionOffset[j], uint2double(keys[offset + j * 2 + 0]), uint2double(keys[offset + j * 2 + 1]));
					if(!matched) {
						break;
					}
				}
			}
			output[i] = matched ? (long) keys[offset + qsize] : -1;
		}
	}
//	
//	#if USE_DYNAMIC_PARALLELISM
//	
//	__global__ static void cudaTestKeyDP(TripKey *keys, KdQuery query, int nSrc, float *srcRegion, int nDst, float *dstRegion, int *output)
//	{
//	  const int index = (blockIdx.x * blockDim.x + threadIdx.x);
//	  bool matched = query.isTimeMatched(keys+index)
//	      && Neighborhoods::isInside(nSrc, srcRegion, keys[index].pickup_lat, keys[index].pickup_long)
//	      && Neighborhoods::isInside(nDst, dstRegion, keys[index].dropoff_lat, keys[index].dropoff_long);
//	  output[index] = matched?(int)keys[index].index:-1;
//	}
//	
//	__global__ static void cudaTestBlockDP(int numBlocks, uint32_t *blockRanges, TripKey *keys, uint32_t *queryRanges, KdQuery query, int nSrc, float *srcRegion, int nDst, float *dstRegion, int *output)
//	{
//	  const int index = (blockIdx.x * blockDim.x + threadIdx.x);
//	  bool matched = (index<numBlocks) && KdQuery::rangeMatched(blockRanges + index*12, queryRanges);
//	  if (matched) {
//	    hipStream_t s;
//	    hipStreamCreateWithFlags(&s, hipStreamNonBlocking);
//	    int nThread = THREAD_N;
//	    cudaTestKeyDP<<<KdBlock::MAX_RECORDS_PER_BLOCK/nThread, nThread, 0, s>>>(keys+index*KdBlock::MAX_RECORDS_PER_BLOCK, query, nSrc, srcRegion, nDst, dstRegion, output+index*KdBlock::MAX_RECORDS_PER_BLOCK);
//	    hipStreamDestroy(s);
//	  }
//	}
//	
//	__global__ static void cudaTestBlockBatchDP(int numBlocks, uint32_t *blockRanges, uint32_t *dBlockResults, TripKey *keys, uint32_t *queryRanges, KdQuery query, int nSrc, float *srcRegion, int nDst, float *dstRegion, int *output)
//	{
//	  __shared__  int sum[THREAD_N+1];
//	  const int index = (blockIdx.x * blockDim.x + threadIdx.x);
//	  const int tid = threadIdx.x;
//	  int offset, i;
//	  
//	  bool matched = (index<numBlocks) && KdQuery::rangeMatched(blockRanges + index*12, queryRanges);  
//	  sum[tid] = matched?1:0;
//	  for (offset=1, i=blockDim.x>>1; i>0; i>>=1) {
//	    __syncthreads();
//	    if(tid<i) {
//	      sum[offset*(2*tid+2)-1] += sum[offset*(2*tid+1)-1];
//	    }
//	    offset <<= 1;
//	  }
//	
//	  __syncthreads();
//	  if (tid==blockDim.x-1) {
//	    sum[blockDim.x] = sum[tid];
//	    sum[tid] = 0;
//	  }
//	
//	  for (i=1; i<blockDim.x; i<<=1) {  
//	    offset >>= 1;  
//	    __syncthreads();  
//	    if (tid<i) {  
//	        int ai = offset*(2*tid+1)-1;
//	        int bi = offset*(2*tid+2)-1;
//	        int temp = sum[ai];  
//	        sum[ai] = sum[bi];  
//	        sum[bi] += temp;
//	    }
//	  }
//	  
//	  __syncthreads();
//	  if (matched)
//	    dBlockResults[blockIdx.x*blockDim.x+sum[tid]] = index;
//	
//	  if (tid==blockDim.x-1) {
//	    int count = sum[blockDim.x]*KdBlock::MAX_RECORDS_PER_BLOCK;
//	    if (count>0) {
//	      hipStream_t s;
//	      int nBlock = (count+THREAD_N*RECORDS_PER_THREAD-1)/(THREAD_N*RECORDS_PER_THREAD);
//	      hipStreamCreateWithFlags(&s, hipStreamNonBlocking);
//	      cudaTestKey<<<nBlock, THREAD_N, 0, s>>>(count, keys, dBlockResults+blockIdx.x*blockDim.x, query, nSrc, srcRegion, nDst, dstRegion,
//	                                              output+blockIdx.x*blockDim.x*KdBlock::MAX_RECORDS_PER_BLOCK);
//	      hipStreamDestroy(s);
//	    }
//	  }
//	}
//	
//	#endif
	
	void testBlock(int numBlocks, uint64_t *dBlocks, uint64_t *dBlockResults, uint64_t *dQueryRanges, hipStream_t dStream, uint32_t size)
	{
	  int nThread = THREAD_N;
	  int nBlock = (numBlocks+nThread-1)/(nThread);
	  cudaTestBlock<<<nBlock, nThread, 0, dStream>>>(dBlocks, numBlocks, dBlockResults, dQueryRanges, size);
	}

	void print_gpu(uint64_t * dBlocks) {
//		  uint64_t tmp=1234;
//		  hipMemcpy(dBlocks, &tmp, sizeof(uint64_t), hipMemcpyHostToDevice);
		  printf("before print\n");
		  cudaprint_gpu<<<1,1>>>(dBlocks);
		  hipDeviceSynchronize();
//		  hipMemcpy(&tmp, dBlocks, sizeof(uint64_t), hipMemcpyDeviceToHost);
//		  printf("after execute %lu\n", tmp);
	}
	void testKey(int numRecords, uint64_t *dBlockResults, TripKey *dKeys, uint32_t qsize, uint64_t * dQueryRange, int noRegions, uint32_t* dRegionSize, 
					uint32_t* dRegionOffset, float *dRegions, long *dResults, hipStream_t dStream) {
		int nThread = THREAD_N;
		int nBlock = (numRecords + nThread * RECORDS_PER_THREAD - 1) / (nThread * RECORDS_PER_THREAD);
//		printf("no blocks: %d %d\n",nBlock,nThread);
		cudaTestKey<<<nBlock, nThread, 0, dStream>>>(numRecords, dKeys, dBlockResults, qsize, dQueryRange, noRegions, dRegionSize, dRegionOffset, dRegions, dResults);
	}
	
	//	void testKeyAll(int numRecords, TripKey *dKeys, const KdQuery &query, int nSrc, float *dSrcRegion, int nDst, float *dDstRegion, int *dResults, hipStream_t dStream)
	void testKeyAll(int numRecords, TripKey *dKeys, uint32_t qsize, uint64_t *dQueryRange, int noRegions, uint32_t* dRegionSize, uint32_t* dRegionOffset, float* dRegions, 
			long* dResults, hipStream_t dStream) {
		int nThread = THREAD_N;
		int nBlock = (numRecords + nThread * RECORDS_PER_THREAD - 1) / (nThread * RECORDS_PER_THREAD);
		cudaTestKeyAll<<<nBlock, nThread, 0, dStream>>>(numRecords, dKeys, qsize, dQueryRange, noRegions, dRegionSize, dRegionOffset, dRegions, dResults);
	}
	
	struct is_valid {
		__host__  __device__
		bool operator()(const int x) {
			return x != -1;
		}
	};
	
	int compactBlocks(int numBlocks, uint64_t *dBlockResults, std::vector<uint64_t> &blockResults) {
		thrust::device_ptr<uint64_t> dpSrc(dBlockResults);
		size_t num = thrust::remove(dpSrc, dpSrc + numBlocks, -1) - dpSrc;
		size_t currentSize = blockResults.size();
		printf("num = %lu, curSize = %lu\n", num, currentSize);
		blockResults.resize(currentSize + num);
		thrust::copy(dpSrc, dpSrc + num, &blockResults[currentSize]);
		return num;
	}
	
	void getResults(int numRecords, long *dResults, std::vector<long> &results,
						hipStream_t dStream) {
		thrust::device_ptr<long> dpResults(dResults);
		int numResults = thrust::remove(dpResults, dpResults + numRecords, -1) - dpResults;
		size_t currentSize = results.size();
		results.resize(currentSize + numResults);
		thrust::copy(dpResults, dpResults + numResults, &results[currentSize]);
	}
	
//	void testBlockDP(int numBlocks, uint32_t *dBlocks, uint32_t *dBlockResults, TripKey *dKeys, uint32_t *dQueryRanges, const KdQuery &query, int nSrc, float *dSrcRegion, int nDst, float *dDstRegion, int *dResults, hipStream_t dStream)
//	{
//	#if USE_DYNAMIC_PARALLELISM
//	  hipMemset(dResults, 0xFF, numBlocks*KdBlock::MAX_RECORDS_PER_BLOCK*sizeof(int));
//	  int nThread = THREAD_N;
//	  int nBlock = (numBlocks+nThread-1)/nThread;
//	  cudaTestBlockBatchDP<<<nBlock, nThread, 0, dStream>>>(numBlocks, dBlocks, dBlockResults, dKeys, dQueryRanges, query, nSrc, dSrcRegion, nDst, dDstRegion, dResults);
//	#else
//	  fprintf(stderr, "FATAL ERROR: USE_DYNAMIC_PARALLELISM is not enabled.\n");
//	  exit(1);
//	#endif
//	}
	
	__device__ int computePos(int size, volatile int *sum, int tid)
	{
	  int offset, i;
	  for (offset=1, i=size>>1; i>0; i>>=1) {
	    __syncthreads();
	    if(tid<i) {
	      sum[offset*(2*tid+2)-1] += sum[offset*(2*tid+1)-1];
	    }
	    offset <<= 1;
	  }
	
	  __syncthreads();
	  int count = sum[size-1];
	  if (tid==0)
	    sum[size-1] = 0;
	
	  for (i=1; i<size; i<<=1) {  
	    offset >>= 1;  
	    __syncthreads();  
	    if (tid<i) {  
	        int ai = offset*(2*tid+1)-1;
	        int bi = offset*(2*tid+2)-1;
	        int temp = sum[ai];  
	        sum[ai] = sum[bi];  
	        sum[bi] += temp;
	    }
	  }
	
	  return count;
	}


}

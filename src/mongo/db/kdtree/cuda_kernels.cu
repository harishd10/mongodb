#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <helper_string.h>
#include "cuda_kernels.hpp"
#include <thrust/copy.h>
#include <thrust/device_ptr.h>
#include <thrust/remove.h>
#include "KdBlock.hpp"

#include <iostream>
#include <cstdio>
#include <cstdlib>

static const int THREAD_N = 128;
static const int RECORDS_PER_THREAD = 8;
//#define USE_DYNAMIC_PARALLELISM 1

namespace mongo {

	__global__ static void cudaprint_gpu(uint64_t *tmp) {
		printf("In gpu \n");
		double * dd = (double *)tmp;
		printf("%lf %lf\n", dd[0], dd[1]);
		printf("%lf %lf\n", dd[2], dd[3]);
		printf("%lf %lf\n", dd[4], dd[5]);
	}
	
	__global__ static void cudaTestBlock(uint64_t *blockRanges, int numBlocks, uint32_t *output, uint64_t *queryRanges, uint32_t size)
	{
		const uint32_t index = (blockIdx.x * blockDim.x + threadIdx.x);
		bool matched = (index < numBlocks) && KdQuery::rangeMatched(blockRanges + index * size * 2, queryRanges, size);
		output[index] = matched ? index : 0xFFFFFFFF;
	}
	
	// called only for in memory
	__global__ static void cudaTestKey(int numRecords, TripKey *keys, uint32_t *blockIndex, uint32_t qsize, uint64_t * queryRange, int noRegions, uint32_t* regionSize, 
						uint32_t* regionOffset, float *regions, int *output)
	{
		const int tid = (blockIdx.x * blockDim.x + threadIdx.x);
		const int stride = gridDim.x * blockDim.x;
		
		for (int i = tid; i < numRecords; i += stride) {
			int offset = KdBlock::MAX_RECORDS_PER_BLOCK
					* blockIndex[i / KdBlock::MAX_RECORDS_PER_BLOCK]
					+ i % KdBlock::MAX_RECORDS_PER_BLOCK;
			offset = offset * (qsize + 1);
			bool matched = KdQuery::queryMatched(queryRange,keys + offset, qsize);
			if(matched) {
				for(int j = 0;j < noRegions;j ++) {
					matched = Neighborhoods::isInside(regionSize[j], regions + regionOffset[j], uint2double(keys[offset + j * 2 + 0]), uint2double(keys[offset + j * 2 + 1]));
					if(!matched) {
						break;
					}
				}
			}
			output[i] = matched ? (int) keys[offset + qsize] : -1;
		}
	}

	
	__global__ static void cudaTestKeyAll(int numRecords, TripKey *keys, uint32_t qsize, uint64_t * queryRange, int noRegions, uint32_t* regionSize, 
						uint32_t* regionOffset, float *regions, long *output)
	{
		const int tid = (blockIdx.x * blockDim.x + threadIdx.x);
		const int stride = gridDim.x * blockDim.x;
		for (int i = tid; i < numRecords; i += stride) {
			int offset = i * (qsize + 1);
			bool matched = KdQuery::queryMatched(queryRange,keys + offset, qsize);
			if(matched) {
				for(int j = 0;j < noRegions;j ++) {
					matched = Neighborhoods::isInside(regionSize[j], regions + regionOffset[j], uint2double(keys[offset + j * 2 + 0]), uint2double(keys[offset + j * 2 + 1]));
					if(!matched) {
						break;
					}
				}
			}
			output[i] = matched ? (long) keys[offset + qsize] : -1;
		}
	}
	
#if USE_DYNAMIC_PARALLELISM

	__global__ static void cudaTestBlockBatchDP(int numBlocks, uint64_t *blockRanges, uint32_t *dBlockResults, TripKey *keys, uint32_t size, 
			uint64_t *queryRanges, int noRegions, uint32_t* regionSize, uint32_t* regionOffset, float *regions, int *output)
	{
		__shared__ int sum[THREAD_N+1];
		const uint32_t index = (blockIdx.x * blockDim.x + threadIdx.x);
		const int tid = threadIdx.x;
		int offset, i;
	
		bool matched = (index < numBlocks) && KdQuery::rangeMatched(blockRanges + index * size * 2, queryRanges, size);
		
		sum[tid] = matched?1:0;
		for (offset=1, i=blockDim.x>>1; i>0; i>>=1) {
			__syncthreads();
			if(tid<i) {
				sum[offset*(2*tid+2)-1] += sum[offset*(2*tid+1)-1];
			}
			offset <<= 1;
		}
	
		__syncthreads();
		if (tid==blockDim.x-1) {
			sum[blockDim.x] = sum[tid];
			sum[tid] = 0;
		}
	
		for (i=1; i<blockDim.x; i<<=1) {
			offset >>= 1;
			__syncthreads();
			if (tid<i) {
				int ai = offset*(2*tid+1)-1;
				int bi = offset*(2*tid+2)-1;
				int temp = sum[ai];
				sum[ai] = sum[bi];
				sum[bi] += temp;
			}
		}
	
		__syncthreads();
		if (matched)
		dBlockResults[blockIdx.x*blockDim.x+sum[tid]] = index;
	
		if (tid==blockDim.x-1) {
			int count = sum[blockDim.x]*KdBlock::MAX_RECORDS_PER_BLOCK;
			if (count>0) {
				hipStream_t s;
				int nBlock = (count+THREAD_N*RECORDS_PER_THREAD-1)/(THREAD_N*RECORDS_PER_THREAD);
				hipStreamCreateWithFlags(&s, hipStreamNonBlocking);
				cudaTestKey<<<nBlock, THREAD_N, 0, s>>>(count, keys, dBlockResults+blockIdx.x*blockDim.x, size, queryRanges, noRegions, regionSize, regionOffset, 
						regions, output+blockIdx.x*blockDim.x*KdBlock::MAX_RECORDS_PER_BLOCK);
				hipStreamDestroy(s);
			}
		}
	}

#endif
	
	void testBlock(int numBlocks, uint64_t *dBlocks, uint32_t *dBlockResults, uint64_t *dQueryRanges, hipStream_t dStream, uint32_t size)
	{
		int nThread = THREAD_N;
		int nBlock = (numBlocks + nThread - 1) / (nThread);
		cudaTestBlock<<<nBlock, nThread, 0, dStream>>>(dBlocks, numBlocks, dBlockResults, dQueryRanges, size);
	}

	void print_gpu(uint64_t * dBlocks) {
		  printf("before print\n");
		  cudaprint_gpu<<<1,1>>>(dBlocks);
		  hipDeviceSynchronize();
	}
	void testKey(int numRecords, uint32_t *dBlockResults, TripKey *dKeys, uint32_t qsize, uint64_t * dQueryRange, int noRegions, uint32_t* dRegionSize, 
					uint32_t* dRegionOffset, float *dRegions, int *dResults, hipStream_t dStream) {
		int nThread = THREAD_N;
		int nBlock = (numRecords + nThread * RECORDS_PER_THREAD - 1) / (nThread * RECORDS_PER_THREAD);
		cudaTestKey<<<nBlock, nThread, 0, dStream>>>(numRecords, dKeys, dBlockResults, qsize, dQueryRange, noRegions, dRegionSize, dRegionOffset, dRegions, dResults);
		checkCudaErrors(hipDeviceSynchronize());
	}
	
	void testKeyAll(int numRecords, TripKey *dKeys, uint32_t qsize, uint64_t *dQueryRange, int noRegions, uint32_t* dRegionSize, uint32_t* dRegionOffset, float* dRegions, 
			long* dResults, hipStream_t dStream) {
		int nThread = THREAD_N;
		int nBlock = (numRecords + nThread * RECORDS_PER_THREAD - 1) / (nThread * RECORDS_PER_THREAD);
		cudaTestKeyAll<<<nBlock, nThread, 0, dStream>>>(numRecords, dKeys, qsize, dQueryRange, noRegions, dRegionSize, dRegionOffset, dRegions, dResults);
	}
	
	struct is_valid {
		__host__  __device__
		bool operator()(const int x) {
			return x != -1;
		}
	};
	
	int compactBlocks(int numBlocks, uint32_t *dBlockResults, std::vector<uint32_t> &blockResults) {
		thrust::device_ptr<uint32_t> dpSrc(dBlockResults);
		size_t num = thrust::remove(dpSrc, dpSrc + numBlocks, -1) - dpSrc;
		size_t currentSize = blockResults.size();
		blockResults.resize(currentSize + num);
		thrust::copy(dpSrc, dpSrc + num, &blockResults[currentSize]);
		return num;
	}
	
	void getResults(int numRecords, long *dResults, std::vector<long> &results,
						hipStream_t dStream) {
		thrust::device_ptr<long> dpResults(dResults);
		int numResults = thrust::remove(dpResults, dpResults + numRecords, -1) - dpResults;
		size_t currentSize = results.size();
		results.resize(currentSize + numResults);
		thrust::copy(dpResults, dpResults + numResults, &results[currentSize]);
	}
	
	void getResults(int numRecords, int* dResults, std::vector<long> &results,
						hipStream_t dStream, int *dTmp) {
		thrust::device_ptr<int> dpResults(dResults);
		thrust::device_ptr<int> dpTmp(dTmp);
		int numResults = thrust::remove_copy(dpResults, dpResults + numRecords, dpTmp, -1) - dpTmp;
		size_t currentSize = results.size();
		results.resize(currentSize + numResults);
		thrust::copy(dpTmp, dpTmp + numResults, &results[currentSize]);
	}
	
	__device__ int computePos(int size, volatile int *sum, int tid)
	{
	  int offset, i;
	  for (offset=1, i=size>>1; i>0; i>>=1) {
	    __syncthreads();
	    if(tid<i) {
	      sum[offset*(2*tid+2)-1] += sum[offset*(2*tid+1)-1];
	    }
	    offset <<= 1;
	  }
	
	  __syncthreads();
	  int count = sum[size-1];
	  if (tid==0)
	    sum[size-1] = 0;
	
	  for (i=1; i<size; i<<=1) {  
	    offset >>= 1;  
	    __syncthreads();  
	    if (tid<i) {  
	        int ai = offset*(2*tid+1)-1;
	        int bi = offset*(2*tid+2)-1;
	        int temp = sum[ai];  
	        sum[ai] = sum[bi];  
	        sum[bi] += temp;
	    }
	  }
	
	  return count;
	}

	void testBlockDP(int numBlocks, uint64_t *dBlocks, uint32_t *dBlockResults, TripKey *dKeys, uint32_t qsize, uint64_t *dQueryRanges, int noRegions,
			uint32_t* dRegionSize, uint32_t* dRegionOffset, float* dRegions, int* dResults, hipStream_t dStream)
	{
		//printf("dp \n");
#if USE_DYNAMIC_PARALLELISM
	  hipMemset(dResults, 0xFFFFFFFF, numBlocks*KdBlock::MAX_RECORDS_PER_BLOCK*sizeof(int));
	  int nThread = THREAD_N;
	  int nBlock = (numBlocks+nThread-1)/nThread;
	  cudaTestBlockBatchDP<<<nBlock, nThread, 0, dStream>>>(numBlocks, dBlocks, dBlockResults, dKeys, qsize, dQueryRanges, noRegions, 
			  dRegionSize, dRegionOffset, dRegions, dResults);
#else
	  fprintf(stderr, "FATAL ERROR: USE_DYNAMIC_PARALLELISM is not enabled.\n");
	  exit(1);
#endif
	}
}
